#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

// 初始化CUDA设备
hipSetDevice(0); // 设置使用的GPU编号，假设为0


__global__ void searchKnn(float *queryVectors,
						  float *databaseVectors,
						  int *indices,
						  float *distances,
						  int vectorSize,
						  int numVectors,
						  int k) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < numVectors) {
	float distance = 0;
	for (int i = 0; i < vectorSize; ++i) {
	  float diff = queryVectors[blockIdx.y * vectorSize + i] - databaseVectors[idx * vectorSize + i];
	  distance += diff * diff;
	}
	distances[idx] = sqrt(distance);
	indices[idx] = idx;
  }
}

__device__ void deviceFunction() {
  float *d_queryVectors, *d_databaseVectors;
  int *d_indices;
  float *d_distances;

  hipMalloc(&d_queryVectors, sizeof(float) * numQueries * vectorSize);
  hipMalloc(&d_databaseVectors, sizeof(float) * numVectors * vectorSize);
  hipMalloc(&d_indices, sizeof(int) * numVectors);
  hipMalloc(&d_distances, sizeof(float) * numVectors);

  hipMemcpy(d_queryVectors, host_queryVectors, sizeof(float) * numQueries * vectorSize, hipMemcpyHostToDevice);
  hipMemcpy(d_databaseVectors, host_databaseVectors, sizeof(float) * numVectors * vectorSize, hipMemcpyHostToDevice);

  dim3 blocks(numVectors);
  dim3 threads(1);
  searchKnn<<<blocks, threads>>>(d_queryVectors, d_databaseVectors, d_indices, d_distances, vectorSize, numVectors, k);

// 把结果从GPU内存复制回主机内存
  int *h_indices = new int[numVectors];
  float *h_distances = new float[numVectors];
  hipMemcpy(h_indices, d_indices, sizeof(int) * numVectors, hipMemcpyDeviceToHost);
  hipMemcpy(h_distances, d_distances, sizeof(float) * numVectors, hipMemcpyDeviceToHost);

// 释放GPU内存
  hipFree(d_queryVectors);
  hipFree(d_databaseVectors);
  hipFree(d_indices);
  hipFree(d_distances);
}